#include "hip/hip_runtime.h"

#include "ldpc_bp_decode.cuh"
#include "ldpc_bp_decode_kernel.cuh"
//#include "driverUtility.h"

#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>

#if USE_TEXTURE_ADDRESS
	hipArray* arr_mcv;
	hipArray* arr_mvc;
	hipChannelFormatDesc channelDesc;
#endif

bool ldpc_gpu::syndrome_check_gpu() 
{
	dim3 block( SIZE_BLOCK );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	syndrome_check_kernel<<< grid, block >>>( d_LLRout, d_sumX2, ncheck, d_V, d_synd );

	int h_synd=0;
	hipMemcpy( &h_synd, d_synd, sizeof(int), hipMemcpyDeviceToHost );

	return h_synd == 0;   // codeword is valid
}

void ldpc_gpu::updateVariableNode_gpu() 
{
	dim3 block( SIZE_BLOCK );
	dim3 grid( (nvar + block.x - 1) / block.x );

	updateVariableNode_kernel<<< grid, block >>>( nvar, ncheck, d_sumX1, d_mcv, d_iind, d_LLRin, d_LLRout, d_mvc );
}

void ldpc_gpu::updateCheckNode_gpu()
{
	dim3 block( SIZE_BLOCK );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	updateCheckNode_kernel<<< grid, block >>>(ncheck, nvar, 
		d_sumX2, d_mvc, d_jind, d_logexp_table, Dint1, Dint2, Dint3,
		QLLR_MAX, d_mcv );	// Shared not faster
}

void ldpc_gpu::initializeMVC_gpu( )
{
	dim3 block( 256 );
	dim3 grid( (nvar + block.x - 1) / block.x );

	initializeMVC_kernel<<< grid, block >>>( nvar, d_sumX1, d_LLRin, d_mvc );
}

int ldpc_gpu::bp_decode(int *LLRin, int *LLRout,
	bool psc /*= true*/,			//!< check syndrom after each iteration
	int max_iters /*= 50*/ )		//!< Maximum number of iterations
{
	hipMemcpy( d_LLRin, LLRin, nvar * sizeof(int), hipMemcpyHostToDevice );

  // initial step
	initializeMVC_gpu();

  bool is_valid_codeword = false;
  int iter = 0;
  do {
    iter++;
    //if (nvar >= 100000) { it_info_no_endl_debug("."); }
    // --------- Step 1: check to variable nodes ----------
	updateCheckNode_gpu();

#if USE_TEXTURE_ADDRESS
    // update the array to the texture
    hipMemcpyToArray(arr_mcv, 0, 0, d_mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyDeviceToDevice);
#endif

    // step 2: variable to check nodes
	updateVariableNode_gpu();

#if USE_TEXTURE_ADDRESS
    // update the array to the texture
    hipMemcpyToArray(arr_mvc, 0, 0, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToDevice);
#endif

#if	USE_TABLE_CODE
	updateConstantMemoryLLRByte( d_LLRout );
#endif

	if (psc && syndrome_check_gpu()) {
	  is_valid_codeword = true;
      break;
    }
  }
  while (iter < max_iters);

  hipMemcpy( LLRout, d_LLRout, nvar * sizeof(int), hipMemcpyDeviceToHost );


  return (is_valid_codeword ? iter : -iter);
}

int ldpc_gpu::bp_decode_once(int *LLRin, char *LLRout,
	bool psc /*= true*/,			//!< check syndrom after each iteration
	int max_iters /*= 50*/ )		//!< Maximum number of iterations
{
	hipMemcpy( d_LLRin, LLRin, nvar * sizeof(int), hipMemcpyHostToDevice );

 	dim3 block( SIZE_BLOCK );
	dim3 grid( (nvar + block.x - 1) / block.x );

	// initial step
	initializeMVC_kernel<<< grid, block >>>( nvar, d_sumX1, d_LLRin, d_mvc );

#if WRITE_FILE_FOR_DRIVER
	static bool bRunOnce1 = false;
	if( !bRunOnce1 ){
		hipMemcpy( h_mvc, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToHost );
		writeArray( h_mvc, nvar * nmaxX1, "../data/mvcInit.txt" );		
		bRunOnce1 = true;
	}
#endif

	int not_valid_codeword = true;
	int iter = 1;
	for( ; iter < max_iters && not_valid_codeword; iter ++ )
	{
		// --------- Step 1: check to variable nodes ----------
		updateCheckNodeOpti_kernel<<< grid, block >>>(ncheck, nvar, 
			d_sumX2, d_mvc, d_jind, d_logexp_table, Dint1, Dint2, Dint3,QLLR_MAX, 
			d_mcv );	// Shared not faster

				
#if WRITE_FILE_FOR_DRIVER
		static bool bRunOnce1 = false;
		if( iter == 1 && !bRunOnce1 ){
			hipMemcpy( h_mcv, d_mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyDeviceToHost );

			writeArray( h_mcv, ncheck * nmaxX2, "../data/mcv.txt" );

			bRunOnce1 = true;
		}
#endif

		// --------- Step 2: variable to check nodes ----------
#if USE_BLOCK_2D
	
	dim3 block( SIZE_BLOCK_2D_X, MAX_VAR_NODE );
	dim3 grid;
	grid.x = (nvar * MAX_VAR_NODE + SIZE_BLOCK_2D_X * MAX_VAR_NODE - 1) 
				/ (SIZE_BLOCK_2D_X * MAX_VAR_NODE) ;

	updateVariableNodeOpti2D_kernel<<< grid, block >>>( nvar, ncheck, 
		d_sumX1, d_mcv, d_iind, d_LLRin, 
		d_LLRout, d_mvc );
#else
		updateVariableNodeOpti_kernel<<< grid, block >>>( nvar, ncheck, 
			d_sumX1, d_mcv, d_iind, d_LLRin, 
			d_LLRout, d_mvc );
#endif

		// --------- Step 3: check syndrome ��żУ�� ----------
#if 0
		syndrome_check_kernel<<< grid, block >>>( d_LLRout, d_sumX2, ncheck, d_V, 
			d_synd );

		hipMemcpy( &not_valid_codeword, d_synd, sizeof(int), hipMemcpyDeviceToHost );
#else
		hipMemcpy( LLRout, d_LLRout, nvar * sizeof(char), hipMemcpyDeviceToHost );
		
#if WRITE_FILE_FOR_DRIVER
		static bool bRunOnce2 = false;
		if( iter == 1 && !bRunOnce2 ){
			hipMemcpy( h_mvc, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToHost );

			writeArray( LLRout, nvar, "../data/output.txt" );
			writeArray( h_mvc, nvar * nmaxX1, "../data/mvc.txt" );		

			bRunOnce2 = true;
		}
#endif

		if (psc && check_parity_cpu(LLRout)) {
			 not_valid_codeword = false;
			break;
		}
#endif
	}
  
  //hipMemcpy( LLRout, d_LLRout, nvar * sizeof(char), hipMemcpyDeviceToHost );


  return (!not_valid_codeword ? iter : -iter);
}

bool ldpc_gpu::check_parity_cpu(char *LLR) 
{
	// Please note the IT++ convention that a sure zero corresponds to
	// LLR=+infinity
	int i, j, synd, vi;

	for (j = 0; j < ncheck; j++) {
		synd = 0;
		int vind = j; // tracks j+i*ncheck
		for (i = 0; i < h_sumX2[j]; i++) {
			vi = h_V[vind];
			if (LLR[vi]) {
				synd++;
			}
			vind += ncheck;
		}
		if ((synd&1) == 1) {
			return false;  // codeword is invalid
		}
	}
	return true;   // codeword is valid
}

bool ldpc_gpu::initialize( int nvar, int ncheck,
	int nmaxX1, int nmaxX2,
	int* sumX1, int* sumX2, int* iind, int* jind, int* V, 	// Parity check matrix parameterization
	int* mvc, int* mcv,	// temporary storage for decoder (memory allocated when codec defined)
	short int Dint1, short int Dint2, short int Dint3,
	int* logexp_table		//! The lookup tables for the decoder
	)
{
	this->nvar = nvar;		this->ncheck = ncheck;
	this->nmaxX1 = nmaxX1;	this->nmaxX2 = nmaxX2; // max(sumX1) max(sumX2)
	this->Dint1 = Dint1;	this->Dint2 = Dint2;	this->Dint3 = Dint3;	//! Decoder (lookup-table) parameters
	
	this->h_V = V;
	this->h_sumX2 = sumX2;

	//max_cnd = 200;
	QLLR_MAX = (std::numeric_limits<int>::max() >> 4);

	hipMalloc( (void**)&d_LLRin, nvar * sizeof(int) );
	hipMalloc( (void**)&d_LLRout, nvar * sizeof(char) );
	hipMemset( d_LLRout, 1, nvar * sizeof(char) );

	hipMalloc( (void**)&d_synd, 1 * sizeof(int) );
	hipMemset( d_synd, 0, 1 * sizeof(int) );
	
	hipMalloc( (void**)&d_sumX1, nvar * sizeof(int) );		// const 64 K
	hipMemcpy( d_sumX1, sumX1, nvar * sizeof(int), hipMemcpyHostToDevice );

	hipMalloc( (void**)&d_sumX2, ncheck * sizeof(int) );	// const 32 K
	hipMemcpy( d_sumX2, sumX2, ncheck * sizeof(int), hipMemcpyHostToDevice );

	hipMalloc( (void**)&d_iind, nvar * nmaxX1 * sizeof(int) );		// const 1.2 M
	hipMemcpy( d_iind, iind, nvar * nmaxX1 * sizeof(int), hipMemcpyHostToDevice );
	
	hipMalloc( (void**)&d_jind, ncheck * nmaxX2 * sizeof(int) );	// const 300 K
	hipMemcpy( d_jind, jind, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );

	hipMalloc( (void**)&d_V, ncheck * nmaxX2 * sizeof(int) );		// const 300 K
	hipMemcpy( d_V, V, ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );
	
	hipMalloc( (void**)&d_mcv, ncheck * nmaxX2 * sizeof(int) );
	hipMemset( d_mcv, 0, ncheck * nmaxX2 * sizeof(int) );
		
	hipMalloc( (void**)&d_mvc, nvar * nmaxX1 * sizeof(int) );
	hipMemset( d_mvc, 0, nvar * nmaxX1 * sizeof(int) );

	hipMalloc( (void**)&d_logexp_table, Dint2 * sizeof(int) );		// const 1.2 K
	hipMemcpy( d_logexp_table, logexp_table, Dint2 * sizeof(int), hipMemcpyHostToDevice );

	initConstantMemoryLogExp(logexp_table);

#if USE_TEXTURE_ADDRESS
	// cuda texture ------------------------------------------------------------------------------------------
	channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
    hipError_t err = hipMallocArray(&arr_mcv, &channelDesc, ncheck, nmaxX2);
    hipMemcpyToArray(arr_mcv, 0, 0, d_mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyDeviceToDevice);

	texMCV.addressMode[0] = hipAddressModeClamp;
	texMCV.addressMode[1] = hipAddressModeClamp;
    texMCV.filterMode = hipFilterModePoint;
    texMCV.normalized = false;

	hipBindTextureToArray(texMCV, arr_mcv, channelDesc);

	hipMallocArray(&arr_mvc, &channelDesc, nvar, nmaxX1);
    hipMemcpyToArray(arr_mvc, 0, 0, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToDevice);
	hipBindTextureToArray(texMVC, arr_mvc, channelDesc);

#endif

	h_mvc = (int*)malloc(nvar * nmaxX1 * sizeof(int));
	h_mcv = (int*)malloc(ncheck * nmaxX2 * sizeof(int));

	return true;
}


bool ldpc_gpu::release()
{
	hipFree( d_LLRin );	hipFree( d_LLRout );
	
	hipFree( d_synd );

	hipFree( d_sumX1 );	hipFree( d_sumX2 );
	
	hipFree( d_iind );		hipFree( d_jind );
	hipFree( d_V );

	hipFree( d_mcv );		hipFree( d_mvc );
	
	hipFree( d_logexp_table );	

	free( h_mvc );	free( h_mcv );

	return true;
}

ldpc_gpu::~ldpc_gpu()
{
	release();
}