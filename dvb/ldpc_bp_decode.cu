#include "hip/hip_runtime.h"

#include "ldpc_bp_decode.cuh"
#include "ldpc_bp_decode_kernel.cuh"
//#include "driverUtility.h"
#include "dvbUtility.h"

#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <iostream>
using namespace std;
#if USE_TEXTURE_ADDRESS
	hipArray* arr_mcv;
	hipArray* arr_mvc;
	hipChannelFormatDesc channelDesc;
#endif

bool ldpc_gpu::syndrome_check_gpu() 
{
	dim3 block( SIZE_BLOCK );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	syndrome_check_kernel<<< grid, block >>>( d_LLRout, d_sumX2, ncheck, d_V, d_synd );

	int h_synd=0;
	hipMemcpy( &h_synd, d_synd, sizeof(int), hipMemcpyDeviceToHost );

	return h_synd == 0;   // codeword is valid
}

void ldpc_gpu::updateVariableNode_gpu() 
{
	dim3 block( SIZE_BLOCK );
	dim3 grid( (nvar + block.x - 1) / block.x );

	updateVariableNode_kernel<<< grid, block >>>( nvar, ncheck, d_sumX1, d_mcv, d_iind, d_LLRin, d_LLRout, d_mvc );
}

void ldpc_gpu::updateCheckNode_gpu()
{
	dim3 block( SIZE_BLOCK );
	dim3 grid( (ncheck + block.x - 1) / block.x );

	updateCheckNode_kernel<<< grid, block >>>(ncheck, nvar, 
		d_sumX2, d_mvc, d_jind, d_logexp_table, Dint1, Dint2, Dint3,
		QLLR_MAX, d_mcv );	// Shared not faster
}

void ldpc_gpu::initializeMVC_gpu( )
{
	dim3 block( 256 );
	dim3 grid( (nvar + block.x - 1) / block.x );

	initializeMVC_kernel<<< grid, block >>>( nvar, d_sumX1, d_LLRin, d_mvc );
}

int ldpc_gpu::bp_decode(int *LLRin, int *LLRout,
	bool psc /*= true*/,			//!< check syndrom after each iteration
	int max_iters /*= 50*/ )		//!< Maximum number of iterations
{
	hipMemcpy( d_LLRin, LLRin, nvar * sizeof(int), hipMemcpyHostToDevice );

  // initial step
	initializeMVC_gpu();

  bool is_valid_codeword = false;
  int iter = 0;
  do {
    iter++;
    //if (nvar >= 100000) { it_info_no_endl_debug("."); }
    // --------- Step 1: check to variable nodes ----------
	updateCheckNode_gpu();

#if USE_TEXTURE_ADDRESS
    // update the array to the texture
    hipMemcpyToArray(arr_mcv, 0, 0, d_mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyDeviceToDevice);
#endif

    // step 2: variable to check nodes
	updateVariableNode_gpu();

#if USE_TEXTURE_ADDRESS
    // update the array to the texture
    hipMemcpyToArray(arr_mvc, 0, 0, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToDevice);
#endif

#if	USE_TABLE_CODE
	updateConstantMemoryLLRByte( d_LLRout );
#endif

	if (psc && syndrome_check_gpu()) {
	  is_valid_codeword = true;
      break;
    }
  }
  while (iter < max_iters);

  hipMemcpy( LLRout, d_LLRout, nvar * sizeof(int), hipMemcpyDeviceToHost );


  return (is_valid_codeword ? iter : -iter);
}

int ldpc_gpu::bp_decode_once(int *LLRin, char *LLRout,
	bool psc /*= true*/,			//!< check syndrom after each iteration
	int max_iters /*= 50*/ )		//!< Maximum number of iterations
{
	hipMemcpy( d_LLRin, LLRin, nvar * sizeof(int), hipMemcpyHostToDevice );

 	dim3 block( SIZE_BLOCK );
	dim3 grid( (nvar + block.x - 1) / block.x );

	// initial step
	initializeMVC_kernel<<< grid, block >>>( nvar, d_sumX1, d_LLRin, d_mvc );

#if WRITE_FILE_FOR_DRIVER
	static bool bRunOnce1 = false;
	if( !bRunOnce1 ){
		hipMemcpy( h_mvc, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToHost );
		writeArray( h_mvc, nvar * nmaxX1, "../data/mvcInit.txt" );		
		bRunOnce1 = true;
	}
#endif

	int not_valid_codeword = true;
	int iter = 1;
	for( ; iter < max_iters && not_valid_codeword; iter ++ )
	{
		// --------- Step 1: check to variable nodes ----------
		updateCheckNodeOpti_kernel<<< grid, block >>>(ncheck, nvar, 
			d_sumX2, d_mvc, d_jind, d_logexp_table, Dint1, Dint2, Dint3,QLLR_MAX, 
			d_mcv );	// Shared not faster

				
#if WRITE_FILE_FOR_DRIVER
		static bool bRunOnce1 = false;
		if( iter == 1 && !bRunOnce1 ){
			hipMemcpy( h_mcv, d_mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyDeviceToHost );

			writeArray( h_mcv, ncheck * nmaxX2, "../data/mcv.txt" );

			bRunOnce1 = true;
		}
#endif

		// --------- Step 2: variable to check nodes ----------
#if USE_BLOCK_2D
	
	dim3 block( SIZE_BLOCK_2D_X, MAX_VAR_NODE );
	dim3 grid;
	grid.x = (nvar * MAX_VAR_NODE + SIZE_BLOCK_2D_X * MAX_VAR_NODE - 1) 
				/ (SIZE_BLOCK_2D_X * MAX_VAR_NODE) ;

	updateVariableNodeOpti2D_kernel<<< grid, block >>>( nvar, ncheck, 
		d_sumX1, d_mcv, d_iind, d_LLRin, 
		d_LLRout, d_mvc );
#else
		updateVariableNodeOpti_kernel<<< grid, block >>>( nvar, ncheck, 
			d_sumX1, d_mcv, d_iind, d_LLRin, 
			d_LLRout, d_mvc );
#endif

		// --------- Step 3: check syndrome ��żУ�� ----------
#if 0
		syndrome_check_kernel<<< grid, block >>>( d_LLRout, d_sumX2, ncheck, d_V, 
			d_synd );

		hipMemcpy( &not_valid_codeword, d_synd, sizeof(int), hipMemcpyDeviceToHost );
#else
		hipMemcpy( LLRout, d_LLRout, nvar * sizeof(char), hipMemcpyDeviceToHost );
		
#if WRITE_FILE_FOR_DRIVER
		static bool bRunOnce2 = false;
		if( iter == 1 && !bRunOnce2 ){
			hipMemcpy( h_mvc, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToHost );

			writeArray( LLRout, nvar, "../data/output.txt" );
			writeArray( h_mvc, nvar * nmaxX1, "../data/mvc.txt" );		

			bRunOnce2 = true;
		}
#endif

		if (psc && check_parity_cpu(LLRout)) {
			 not_valid_codeword = false;
			break;
		}
#endif
	}
  
  //hipMemcpy( LLRout, d_LLRout, nvar * sizeof(char), hipMemcpyDeviceToHost );


  return (!not_valid_codeword ? iter : -iter);
}

bool ldpc_gpu::check_parity_cpu(char *LLR) 
{
	// Please note the IT++ convention that a sure zero corresponds to
	// LLR=+infinity
	int i, j, synd, vi;

	for (j = 0; j < ncheck; j++) {
		synd = 0;
		int vind = j; // tracks j+i*ncheck
		for (i = 0; i < h_sumX2[j]; i++) {
			vi = h_V[vind];
			if (LLR[vi]) {
				synd++;
			}
			vind += ncheck;
		}
		if ((synd&1) == 1) {
			return false;  // codeword is invalid
		}
	}
	return true;   // codeword is valid
}

bool ldpc_gpu::initialize( )
{
	itpp::LDPC_Generator_Systematic G; // for codes created with ldpc_gen_codes since generator exists
	
	ldpc.load_code(FILENAME_IT, &G);


	int nmaxX1 = max(ldpc.sumX1._data(), ldpc.sumX1.size());
	int nmaxX2 = max(ldpc.sumX2._data(), ldpc.sumX2.size());
	int nminX1 = min(ldpc.sumX1._data(), ldpc.sumX1.size());
	int nminX2 = min(ldpc.sumX2._data(), ldpc.sumX2.size());

	int nmaxI = max(ldpc.iind._data(), ldpc.iind.size());
	int nmaxJ = max(ldpc.jind._data(), ldpc.jind.size());
	int nminI = min(ldpc.iind._data(), ldpc.iind.size());
	int nminJ = min(ldpc.jind._data(), ldpc.jind.size());

#if 1
	cout << "max(iind) = " << nmaxI << endl;// max(iind) = nvar*nmaxX1-1
	cout << "max(jind) = " << nmaxJ << endl;// max(jind) = nvar*nmaxX1-1
	cout << "min(iind) = " << nminI << endl;// min(iind) = 0
	cout << "min(jind) = " << nminJ << endl;// min(jind) = 0

	cout << "ldpc.nvar = " << ldpc.nvar << endl;		// nvar = 16200
	cout << "ldpc.ncheck = " << ldpc.ncheck << endl;	// ncheck = 8100//8073 
	cout << "ldpc.sumX1.size() = " << ldpc.sumX1.size() << endl;	// = nvar
	cout << "ldpc.sumX2.size() = " << ldpc.sumX2.size() << endl;	// = ncheck
	cout << "max(sumX1) = " << nmaxX1 << endl;// max(sumX1) = 3//19
	cout << "max(sumX2) = " << nmaxX2 << endl;// max(sumX2) = 6//10
	cout << "min(sumX1) = " << nminX1 << endl;// min(sumX1) = 3//2
	cout << "min(sumX2) = " << nminX2 << endl;// min(sumX2) = 6//7
	cout << "ldpc.V.size() = " << ldpc.V.size() << endl;			// = ncheck * max(sumX2)
	cout << "ldpc.iind.size() = " << ldpc.iind.size() << endl;		// = nvar * max(sumX1)
	cout << "ldpc.jind.size() = " << ldpc.jind.size() << endl;		// = ncheck * max(sumX2)

	cout << "ldpc.mvc.size() = " << ldpc.mvc.size() << endl;		// = nvar * max(sumX1)
	cout << "ldpc.mcv.size() = " << ldpc.mcv.size() << endl;		// = ncheck * max(sumX2)

	cout << "ldpc.llrcalc.Dint1 = " << ldpc.llrcalc.Dint1 << endl;	// Dint1 = 12
	cout << "ldpc.llrcalc.Dint2 = " << ldpc.llrcalc.Dint2 << endl;	// Dint2 = 300
	cout << "ldpc.llrcalc.Dint3 = " << ldpc.llrcalc.Dint3 << endl;	// Dint3 = 7

	cout << "ldpc.llrcalc.logexp_table.size() = " << ldpc.llrcalc.logexp_table.size() << endl;// = 300
#endif


	this->nvar = ldpc.nvar;		this->ncheck = ldpc.ncheck;
	this->nmaxX1 = nmaxX1;	this->nmaxX2 = nmaxX2; // max(sumX1) max(sumX2)
	this->Dint1 = ldpc.llrcalc.Dint1;	
	this->Dint2 = ldpc.llrcalc.Dint2;	
	this->Dint3 = ldpc.llrcalc.Dint3;	//! Decoder (lookup-table) parameters
	
	this->h_V = ldpc.V._data();
	this->h_sumX2 = ldpc.sumX2._data();

	//max_cnd = 200;
	QLLR_MAX = (1<<31 -1)>>4;//(std::numeric_limits<int>::max() >> 4);

	hipMalloc( (void**)&d_LLRin, nvar * sizeof(int) );
	hipMalloc( (void**)&d_LLRout, nvar * sizeof(char) );
	hipMemset( d_LLRout, 1, nvar * sizeof(char) );

	hipMalloc( (void**)&d_synd, 1 * sizeof(int) );
	hipMemset( d_synd, 0, 1 * sizeof(int) );
	
	hipMalloc( (void**)&d_sumX1, nvar * sizeof(int) );		// const 64 K
	hipMemcpy( d_sumX1, ldpc.sumX1._data(), nvar * sizeof(int), hipMemcpyHostToDevice );

	hipMalloc( (void**)&d_sumX2, ncheck * sizeof(int) );	// const 32 K
	hipMemcpy( d_sumX2, ldpc.sumX2._data(), ncheck * sizeof(int), hipMemcpyHostToDevice );

	hipMalloc( (void**)&d_iind, nvar * nmaxX1 * sizeof(int) );		// const 1.2 M
	hipMemcpy( d_iind, ldpc.iind._data(), nvar * nmaxX1 * sizeof(int), hipMemcpyHostToDevice );
	
	hipMalloc( (void**)&d_jind, ncheck * nmaxX2 * sizeof(int) );	// const 300 K
	hipMemcpy( d_jind, ldpc.jind._data(), ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );

	hipMalloc( (void**)&d_V, ncheck * nmaxX2 * sizeof(int) );		// const 300 K
	hipMemcpy( d_V, ldpc.V._data(), ncheck * nmaxX2 * sizeof(int), hipMemcpyHostToDevice );
	
	hipMalloc( (void**)&d_mcv, ncheck * nmaxX2 * sizeof(int) );
	hipMemset( d_mcv, 0, ncheck * nmaxX2 * sizeof(int) );
		
	hipMalloc( (void**)&d_mvc, nvar * nmaxX1 * sizeof(int) );
	hipMemset( d_mvc, 0, nvar * nmaxX1 * sizeof(int) );

	hipMalloc( (void**)&d_logexp_table, Dint2 * sizeof(int) );		// const 1.2 K
	hipMemcpy( d_logexp_table, ldpc.llrcalc.logexp_table._data(), Dint2 * sizeof(int), hipMemcpyHostToDevice );

	initConstantMemoryLogExp(ldpc.llrcalc.logexp_table._data());

#if USE_TEXTURE_ADDRESS
	// cuda texture ------------------------------------------------------------------------------------------
	channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
    hipError_t err = hipMallocArray(&arr_mcv, &channelDesc, ncheck, nmaxX2);
    hipMemcpyToArray(arr_mcv, 0, 0, d_mcv, ncheck * nmaxX2 * sizeof(int), hipMemcpyDeviceToDevice);

	texMCV.addressMode[0] = hipAddressModeClamp;
	texMCV.addressMode[1] = hipAddressModeClamp;
    texMCV.filterMode = hipFilterModePoint;
    texMCV.normalized = false;

	hipBindTextureToArray(texMCV, arr_mcv, channelDesc);

	hipMallocArray(&arr_mvc, &channelDesc, nvar, nmaxX1);
    hipMemcpyToArray(arr_mvc, 0, 0, d_mvc, nvar * nmaxX1 * sizeof(int), hipMemcpyDeviceToDevice);
	hipBindTextureToArray(texMVC, arr_mvc, channelDesc);

#endif

	h_mvc = (int*)malloc(nvar * nmaxX1 * sizeof(int));
	h_mcv = (int*)malloc(ncheck * nmaxX2 * sizeof(int));

	return true;
}


bool ldpc_gpu::release()
{
	hipFree( d_LLRin );	hipFree( d_LLRout );
	
	hipFree( d_synd );

	hipFree( d_sumX1 );	hipFree( d_sumX2 );
	
	hipFree( d_iind );		hipFree( d_jind );
	hipFree( d_V );

	hipFree( d_mcv );		hipFree( d_mvc );
	
	hipFree( d_logexp_table );	

	free( h_mvc );	free( h_mcv );

	return true;
}

ldpc_gpu::~ldpc_gpu()
{
	release();
}

int ldpc_gpu::bp_decode_once( itpp::vec& softbits, char *LLRout )
{
	itpp::QLLRvec llrIn = ldpc.get_llrcalc().to_qllr(softbits);

	return bp_decode_once( llrIn._data(), LLRout);	
}

int ldpc_gpu::bp_decode_once( double* softbits, char *LLRout )
{
	itpp::vec  softVec( nvar );
	convertBufferToVec( softbits, softVec );
	return bp_decode_once( softVec, LLRout );
}

float ldpc_gpu::get_rate()
{
	return ldpc.get_rate();
}